#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <cmath>
#include <cstddef>
#include <hip/hip_runtime.h>
#include "particles.cuh"
// #include "../include/n_list.cuh"

void writeVTKFile(int step, int num_particles, Particle3D* particles) {
    std::ofstream simulationFile("simulation_" + std::to_string(step) + ".vtk");

    simulationFile << "# vtk DataFile Version 3.0 \n";
    simulationFile << "Lennard-Jones particle simulation \n";
    simulationFile << "ASCII \n";
    simulationFile << "DATASET UNSTRUCTURED_GRID \n";
    simulationFile << "POINTS " << num_particles << " float \n";

    for (int i = 0; i < num_particles; i++) {
        float3 pos = particles[i].getPosition();
        simulationFile << pos.x << " " << pos.y << " " << pos.z << "\n";
    }

    simulationFile << "CELLS " << "0" << " " << "0" << "\n";
    simulationFile << "CELL_TYPES " << "0" << "\n";
    simulationFile << "POINT_DATA " << num_particles << "\n";
    simulationFile << "SCALARS mass float \n";
    simulationFile << "LOOKUP_TABLE default \n";

    for (int i = 0; i < num_particles; i++) {
        simulationFile << particles[i].getMass() << "\n";
    }

    simulationFile << "SCALARS radius float \n";
    simulationFile << "LOOKUP_TABLE default \n";

    for (int i = 0; i < num_particles; i++) {
        simulationFile << particles[i].getRadius() << "\n";
    }

    simulationFile << "VECTORS velocity float \n";
    for (int i = 0; i < num_particles; i++) {
        float3 vel = particles[i].getVelocity();
        simulationFile << vel.x << " " << vel.y << " " << vel.z << "\n";
    }
}


void start_particle_simulation(int time_steps, float step_size, int num_particles, float eps, float sigma, float k_n, float gamma, float gravity,float box_extension, float cut_off_radious)
{
    float3 *forces;
    Particle3D* particles;

    // hipGraph_t graph;
    // std::vector<hipGraphNode_t> nodeDependencies;
    // hipGraphNode_t memcpyNode, kernelNode, memsetNode;

    
    int deviceId;
    hipDeviceProp_t prop;

    float num_cell_1d = box_extension / cut_off_radious;
    float num_cell_total = powf(num_cell_1d, 3);

    hipGetDevice(&deviceId);

    
    hipGetDeviceProperties(&prop, deviceId);

    int numberOfThreads = 256;
    int numberOfBlocks = 32 * prop.multiProcessorCount;

    hipMallocManaged(&particles, num_particles * sizeof(Particle3D));
    hipMemPrefetchAsync(particles,  num_particles * sizeof(Particle3D), deviceId);

    hipMallocManaged(&forces, num_particles * sizeof(float3));
    hipMemPrefetchAsync(forces, num_particles * sizeof(float3), deviceId);

    t_neighbourList *nb_list = nullptr;

    int pos = 5;    
    
    for (int i = 0; i < num_particles; ++i) {
        float x = fmod(pos, box_extension) ;
        float y = (pos * 2 >= box_extension * box_extension) ? fmod(floor((pos * 2) / box_extension), box_extension) : 0;
        float z = (pos * 4 >= box_extension * box_extension) ? fmod(floor((pos * 4) / (box_extension * box_extension) ), box_extension) : 0;

        // if(i == 3){
        //     particles[i] = Particle3D(float3{ x, y, z }, float3{ 2.0f, 0.0f, 0.0f }, 1.0f, 1.0f, nullptr, i);
        // }
        // else if(i == 4){
        //     particles[i] = Particle3D(float3{ x, y, z }, float3{ 0.0f, -2.0f, 0.0f }, 1.0f, 1.0f, nullptr, i);
        // }
        // else
        //     particles[i] = Particle3D(float3{ x, y, z }, float3{ 0.0f, 0.0f, 0.0f }, 1.0f, 1.0f, nullptr, i);

        forces[i] = float3{ 0.0f, 0.0f, 0.0f };

        particles[i] = Particle3D(float3{ x, y, z }, float3{ 0.0f, 0.0f, 0.0f }, 1.0f, 1.0f, nullptr, i);

        pos += 2;
    }

    std::cout << num_particles << ", ";
    

//    writeVTKFile(0, num_particles, particles);
    int step = 0;

    for (int step = 0; step < time_steps; ++step) {
        hipMemset(forces, 0, num_particles * sizeof(float3));
        
        // Compute forces using CUDA
        compute_force_between_particles <<< numberOfBlocks, numberOfThreads >>> (particles, forces, num_particles, eps, sigma, k_n, gamma, gravity, box_extension, cut_off_radious, nb_list);
        
        hipDeviceSynchronize();

        // Integrate particles using CUDA
        apply_integrator_for_particle_euler <<< numberOfBlocks, numberOfThreads >>> (particles, forces, num_particles, step_size, box_extension);
        hipDeviceSynchronize();

        // Write the VTK file
        // writeVTKFile(step + 1, num_particles, particles);

    }

    hipFree(particles);
    hipFree(forces);
}
