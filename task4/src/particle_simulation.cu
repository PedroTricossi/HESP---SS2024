#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <hip/hip_runtime.h>
#include <>
#include "../include/particles.cuh"

void writeVTKFile(int step, int num_particles, Particle3D* particles) {
    std::ofstream simulationFile("simulation_" + std::to_string(step) + ".vtk");

    simulationFile << "# vtk DataFile Version 3.0 \n";
    simulationFile << "Lennard-Jones particle simulation \n";
    simulationFile << "ASCII \n";
    simulationFile << "DATASET UNSTRUCTURED_GRID \n";
    simulationFile << "POINTS " << num_particles << " float \n";

    for (int i = 0; i < num_particles; i++) {
        float3 pos = particles[i].getPosition();
        simulationFile << pos.x << " " << pos.y << " " << pos.z << "\n";
    }

    simulationFile << "CELLS " << "0" << " " << "0" << "\n";
    simulationFile << "CELL_TYPES " << "0" << "\n";
    simulationFile << "POINT_DATA " << num_particles << "\n";
    simulationFile << "SCALARS mass float \n";
    simulationFile << "LOOKUP_TABLE default \n";

    for (int i = 0; i < num_particles; i++) {
        simulationFile << particles[i].getMass() << "\n";
    }

    simulationFile << "VECTORS velocity float \n";
    for (int i = 0; i < num_particles; i++) {
        float3 vel = particles[i].getVelocity();
        simulationFile << vel.x << " " << vel.y << " " << vel.z << "\n";
    }
}


void start_particle_simulation(int time_steps, float step_size, int num_particles, float eps, float sigma, float box_extension){
    int deviceId;
    float3* forces;
    Particle3D* particles;
        
    hipGetDevice(&deviceId);

    int numberOfThreads = 256;
    int numberOfBlocks = 32;

    hipMallocManaged(&particles, num_particles * sizeof(Particle3D));
    hipMallocManaged(&forces, num_particles * sizeof(float3));
    
    std::cout << "BOX: " << box_extension << std::endl;
    
    for (int i = 0; i < num_particles; ++i) {
        float x = fmod(i, 10);
        float y = (i >= 10) ? fmod(floor(i / 10), 10): 0;
        float z = (i >= 100) ? fmod(floor(i / 100), 10) : 0;
        particles[i] = Particle3D(float3{ x, y, z }, float3{ 0.0f, 0.0f, 0.0f }, 1.0f);
        forces[i] = float3{ 0.0f, 0.0f, 0.0f };
    }

    writeVTKFile(0, num_particles, particles);

    for (int step = 0; step < time_steps; ++step) {
        // Reset forces
        hipMemset(forces, 0, num_particles * sizeof(float3));

        // Compute forces using CUDA
        compute_force_between_particles <<< numberOfBlocks, numberOfThreads >>> (particles, forces, num_particles, eps, sigma, box_extension);
        hipDeviceSynchronize();

        // Integrate particles using CUDA
        apply_integrator_for_particle <<< numberOfBlocks, numberOfThreads >>> (particles, forces, num_particles, step_size, box_extension);
        hipDeviceSynchronize();

        // Write the VTK file
        writeVTKFile(step + 1, num_particles, particles);
    }

    hipFree(particles);
    hipFree(forces);
}

