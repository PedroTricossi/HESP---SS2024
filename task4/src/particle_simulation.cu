#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <cmath>
#include <cstddef>
#include <hip/hip_runtime.h>
#include <>
#include "../include/particles.cuh"
#include "../include/n_list.cuh"

void writeVTKFile(int step, int num_particles, Particle3D* particles) {
    std::ofstream simulationFile("simulation_" + std::to_string(step) + ".vtk");

    simulationFile << "# vtk DataFile Version 3.0 \n";
    simulationFile << "Lennard-Jones particle simulation \n";
    simulationFile << "ASCII \n";
    simulationFile << "DATASET UNSTRUCTURED_GRID \n";
    simulationFile << "POINTS " << num_particles << " float \n";

    for (int i = 0; i < num_particles; i++) {
        float3 pos = particles[i].getPosition();
        simulationFile << pos.x << " " << pos.y << " " << pos.z << "\n";
    }

    simulationFile << "CELLS " << "0" << " " << "0" << "\n";
    simulationFile << "CELL_TYPES " << "0" << "\n";
    simulationFile << "POINT_DATA " << num_particles << "\n";
    simulationFile << "SCALARS mass float \n";
    simulationFile << "LOOKUP_TABLE default \n";

    for (int i = 0; i < num_particles; i++) {
        simulationFile << particles[i].getMass() << "\n";
    }

    simulationFile << "VECTORS velocity float \n";
    for (int i = 0; i < num_particles; i++) {
        float3 vel = particles[i].getVelocity();
        simulationFile << vel.x << " " << vel.y << " " << vel.z << "\n";
    }
}


void start_particle_simulation(int time_steps, float step_size, int num_particles, float eps, float sigma, float box_extension, float cut_off_radious){
    float3* forces;
    Particle3D* particles;

    int numberOfThreads = 256;
    int numberOfBlocks = 32 * 46;

    hipMallocManaged(&particles, num_particles * sizeof(Particle3D));
    hipMallocManaged(&forces, num_particles * sizeof(float3));

    t_neighbourList *nb_list = init_neighbourList(box_extension, cut_off_radious);

    
    
    for (int i = 0; i < num_particles; ++i) {
        float x = fmod(i * 2, 10) ;
        float y = (i * 2 >= 10) ? fmod(floor(i * 4 / 10), 10): 0;
        float z = (i * 4 >= 100) ? fmod(floor(i * 8 / 100), 10) : 0;
        particles[i] = Particle3D(float3{ x, y, z }, float3{ 0.0f, 0.0f, 0.0f }, 1.0f, nullptr, i);
        forces[i] = float3{ 0.0f, 0.0f, 0.0f };
    }

    

    for(int i = 0; i < num_particles; i++){
        add_particle(nb_list, &particles[i], cut_off_radious, box_extension);
    }

    std::cout << "num_particles: " << num_particles << std::endl;
    

    writeVTKFile(0, num_particles, particles);
    int step = 0;

    for (int step = 0; step < time_steps; ++step) {
        // Reset forces
        hipMemset(forces, 0, num_particles * sizeof(float3));

        // std::cout << "particle: " << particles[1].getPosition().x << " " << particles[1].getPosition().y << " " << particles[1].getPosition().z << std::endl;

        // Compute forces using CUDA
        compute_force_between_particles <<< numberOfBlocks, numberOfThreads >>> (particles, forces, num_particles, eps, sigma, box_extension, cut_off_radious, nb_list);
        hipDeviceSynchronize();

        // Integrate particles using CUDA
        apply_integrator_for_particle <<< numberOfBlocks, numberOfThreads >>> (particles, forces, num_particles, step_size, box_extension);
        hipDeviceSynchronize();

        // Write the VTK file
        writeVTKFile(step + 1, num_particles, particles);

        // Clean the neighbour list
        clean_particle(nb_list);

        nb_list = init_neighbourList(box_extension, cut_off_radious);

        for(int i = 0; i < num_particles; i++){
            add_particle(nb_list, &particles[i], cut_off_radious, box_extension);
        }
    }

    hipFree(particles);
    hipFree(forces);
}

