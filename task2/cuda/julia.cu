#include "hip/hip_runtime.h"
#include "julia.cuh"
#include <iostream>
#include <cuda/std/complex>
#include <hip/hip_runtime.h>
#include <>

using complex_t = cuda::std::complex<double>;

__global__ void computeJuliaSet(unsigned char* image)
{
    int x_index = threadIdx.x + blockDim.x * blockIdx.x;
    int x_stride = blockDim.x * gridDim.x;

    int y_inedex = blockIdx.y * blockDim.y + threadIdx.y;
    int y_stride = blockDim.y * gridDim.y;

    complex_t Julia_C(-0.77, 0.20); // Julia set constant

    for (int i = x_index; i < WIDTH; i += x_stride) {
        for (int j = y_inedex; j < HEIGHT; j += y_stride) {

            double real = (double)i / WIDTH * 4 - 2;
            double imag = (double)j / HEIGHT * 4 - 2;

            cuda::std::complex<double> z0(real, imag);
            cuda::std::complex<double> z = z0;

            int iterations = 0;

            while (abs(z) <= THRESHOLD && iterations < MAX_ITER)
             {
                z = z * z + Julia_C;
                iterations++;
             }

            unsigned char color = (iterations == MAX_ITER) ? 0 : (unsigned char)(255 * iterations / MAX_ITER);

            image[(j * WIDTH + i)] = color; // Red channel
            image[(j * WIDTH + i) + 1] = color; // Green channel
            image[(j * WIDTH + i) + 2] = color; // Blue channel
        }
    }
}

// Host function to compute Julia set
unsigned char* computeJuliaSetHost() {
    int deviceId;
    hipGetDevice(&deviceId);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, deviceId);

    int size = WIDTH * HEIGHT * 3 * sizeof(unsigned char);

    unsigned char* image;
    hipError_t allocError = hipMallocManaged(&image, size);

     if (allocError != hipSuccess) {
        printf("Error allocating memory: %s\n", hipGetErrorString(allocError));
        return nullptr;
    }

    int numberOfBlocks = 32 * prop.multiProcessorCount; // Declare and initialize numberOfBlocks
    int numberOfThreads = 256; // Declare and initialize numberOfThreads

    computeJuliaSet<<<numberOfBlocks, numberOfThreads>>>(image);
    hipDeviceSynchronize();

    hipError_t launchError = hipGetLastError();

    if (launchError != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(launchError));
        hipFree(image);
        return nullptr;
    }

    return image;
}
