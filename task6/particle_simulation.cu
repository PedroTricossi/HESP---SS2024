#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <fstream>
#include <cmath>
#include <cstddef>
#include <hip/hip_runtime.h>
#include "particles.cuh"
// #include "../include/n_list.cuh"

void writeVTKFile(int step, int num_particles, Particle3D* particles_steam_1, Particle3D* particles_steam_2) {
    std::ofstream simulationFile("simulation_" + std::to_string(step) + ".vtk");

    simulationFile << "# vtk DataFile Version 3.0 \n";
    simulationFile << "Lennard-Jones particle simulation \n";
    simulationFile << "ASCII \n";
    simulationFile << "DATASET UNSTRUCTURED_GRID \n";
    simulationFile << "POINTS " << num_particles  << " float \n";

    for (int i = 0; i < (num_particles / 2); i++) {
        float3 pos = particles_steam_1[i].getPosition();
        simulationFile << pos.x << " " << pos.y << " " << pos.z << "\n";
    }

    for (int i = 0; i < (num_particles / 2); i++) {
        float3 pos = particles_steam_2[i].getPosition();
        simulationFile << pos.x << " " << pos.y << " " << pos.z << "\n";
    }

    simulationFile << "CELLS " << "0" << " " << "0" << "\n";
    simulationFile << "CELL_TYPES " << "0" << "\n";
    simulationFile << "POINT_DATA " << (num_particles) << "\n";
    simulationFile << "SCALARS mass float \n";
    simulationFile << "LOOKUP_TABLE default \n";

    for (int i = 0; i < (num_particles / 2); i++) {
        simulationFile << particles_steam_1[i].getMass() << "\n";
    }

    for (int i = 0; i < (num_particles / 2); i++) {
        simulationFile << particles_steam_2[i].getMass() << "\n";
    }

    simulationFile << "SCALARS radius float \n";
    simulationFile << "LOOKUP_TABLE default \n";

    for (int i = 0; i < (num_particles / 2); i++) {
        simulationFile << particles_steam_1[i].getRadius() << "\n";
    }

    for (int i = 0; i < (num_particles / 2); i++) {
        simulationFile << particles_steam_2[i].getRadius() << "\n";
    }

    simulationFile << "VECTORS velocity float \n";
    for (int i = 0; i < (num_particles / 2); i++) {
        float3 vel = particles_steam_1[i].getVelocity();
        simulationFile << vel.x << " " << vel.y << " " << vel.z << "\n";
    }

    for (int i = 0; i < (num_particles / 2); i++) {
        float3 vel = particles_steam_2[i].getVelocity();
        simulationFile << vel.x << " " << vel.y << " " << vel.z << "\n";
    }
}


void start_particle_simulation(int time_steps, float step_size, int num_particles, float eps, float sigma, float k_n, float gamma, float gravity,float box_extension, float cut_off_radious)
{
    float3 *ha_forces;
    float3 *hb_forces;
    float3 *da_forces;
    float3 *db_forces;
    Particle3D* particles;
    int deviceId;
    hipDeviceProp_t prop;

    float num_cell_1d = box_extension / cut_off_radious;
    float num_cell_total = powf(num_cell_1d, 3);

    hipGetDevice(&deviceId);

    hipGetDeviceProperties(&prop, deviceId);

    int numberOfThreads = 256;
    int numberOfBlocks = 32 * prop.multiProcessorCount;


    // hipMallocManaged(&particles, num_particles * sizeof(Particle3D));
    // hipMemPrefetchAsync(particles,  num_particles * sizeof(Particle3D), deviceId);

    // hipMallocManaged(&forces, num_particles * sizeof(float3));
    // hipMemPrefetchAsync(forces, num_particles * sizeof(float3), deviceId);

    Particle3D *h_a = NULL;
	Particle3D *h_b = NULL;

	hipHostMalloc((void**)&h_a, sizeof(Particle3D) * (num_particles / 2));
	hipHostMalloc((void**)&h_b, sizeof(Particle3D) * (num_particles / 2));
    hipHostMalloc((void**)&ha_forces, sizeof(float3) * (num_particles / 2));
    hipHostMalloc((void**)&hb_forces, sizeof(float3) * (num_particles / 2));

    Particle3D* d_a = NULL; 
	Particle3D* d_b = NULL; 

	hipMalloc((void**)&d_a, sizeof(Particle3D) * (num_particles / 2));
	hipMalloc((void**)&d_b, sizeof(Particle3D) * (num_particles / 2));
    hipMalloc((void**)&da_forces, sizeof(float3) * (num_particles / 2));
    hipMalloc((void**)&db_forces, sizeof(float3) * (num_particles / 2));

    hipStream_t stream[2];
    for (int i = 0; i < 2; i++) {
        hipStreamCreate(&stream[i]);
    }

    t_neighbourList *nb_list = nullptr;

    int pos = 4;    
    
    for (int i = 0; i < num_particles; ++i) {
        float x = fmod(pos, box_extension) ;
        float y = (pos >= box_extension) ? fmod(floor(pos * 2 / box_extension), box_extension): 0;
        float z = (pos * 4 >= box_extension * box_extension) ? fmod(floor((pos * 4) / (box_extension * box_extension) ), box_extension) : 0;
        
        if(i < (num_particles / 2))
            h_a[i] = Particle3D(float3{ x, y, z }, float3{ 0.0f, 0.0f, 0.0f }, 1.0f, 1.0f, nullptr, i);
            
            // if (i == 42) {
            //     h_a[i] = Particle3D(float3{ x, y, z }, float3{ -2.0f, 0.0f, 0.0f }, 1.0f, 1.0f, nullptr, i);
            // }
        else
            h_b[i - (num_particles / 2)] = Particle3D(float3{ x, y, z }, float3{ 0.0f, 0.0f, 0.0f }, 1.0f, 1.0f, nullptr, i);
        
        if (i < num_particles / 2)
            ha_forces[i] = float3{ 0.0f, 0.0f, 0.0f };
        else
            hb_forces[i - (num_particles / 2)] = float3{ 0.0f, 0.0f, 0.0f };

        pos += 2;
    }

    std::cout << num_particles << ", ";

    writeVTKFile(0, num_particles, h_a, h_b);

    hipMemcpyAsync(d_a, h_a, sizeof(Particle3D)* (num_particles / 2), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(d_b, h_b, sizeof(Particle3D)* (num_particles / 2), hipMemcpyHostToDevice, stream[1]);

    hipMemcpyAsync(da_forces, ha_forces, sizeof(float3)* (num_particles / 2), hipMemcpyHostToDevice, stream[0]);
	hipMemcpyAsync(db_forces, hb_forces, sizeof(float3)* (num_particles / 2), hipMemcpyHostToDevice, stream[1]);


    for (int step = 0; step < time_steps; ++step) {
        hipMemset(da_forces, 0, sizeof(float3)* (num_particles / 2));
        hipMemset(db_forces, 0, sizeof(float3)* (num_particles / 2));

        // Compute forces using CUDA

        compute_force_between_particles <<< numberOfBlocks, numberOfThreads, 0, stream[0]>>> (d_a, da_forces, num_particles, eps, sigma, k_n, gamma, gravity, box_extension, cut_off_radious, nb_list);
        compute_force_between_particles <<< numberOfBlocks, numberOfThreads, 0, stream[1]>>> (d_b, db_forces, num_particles, eps, sigma, k_n, gamma, gravity, box_extension, cut_off_radious, nb_list);
        hipDeviceSynchronize();

        compute_colision_between_streams <<< numberOfBlocks, numberOfThreads>>> (d_a, d_b, da_forces, db_forces, num_particles, k_n, gamma, box_extension, cut_off_radious);
        hipDeviceSynchronize();

                
        // Integrate particles using CUDA
        apply_integrator_for_particle_euler <<< numberOfBlocks, numberOfThreads, 0, stream[0]>>> (d_a, da_forces, num_particles, step_size, box_extension);
        apply_integrator_for_particle_euler <<< numberOfBlocks, numberOfThreads, 0, stream[1]>>> (d_b, db_forces, num_particles, step_size, box_extension);
        hipDeviceSynchronize();

        hipMemcpyAsync(h_a, d_a, sizeof(Particle3D)* (num_particles / 2), hipMemcpyDeviceToHost, stream[0]);
	    hipMemcpyAsync(h_b, d_b, sizeof(Particle3D)* (num_particles / 2), hipMemcpyDeviceToHost, stream[1]);

        hipDeviceSynchronize();

        // Write the VTK file
        writeVTKFile(step + 1, num_particles, h_a, h_b);
        // writeVTKFile(0, num_particles, h_b);
    }

    // hipFree(particles);
    // hipFree(forces);
}

/*particle simulation using graphs
void start_particle_simulation(int time_steps, float step_size, int num_particles, float eps, float sigma, float k_n, float gamma, float gravity, float box_extension, float cut_off_radious)
{
    float3 *ha_forces;
    float3 *hb_forces;
    float3 *da_forces;
    float3 *db_forces;
    Particle3D* particles;
    int deviceId;
    hipDeviceProp_t prop;

    float num_cell_1d = box_extension / cut_off_radious;
    float num_cell_total = powf(num_cell_1d, 3);

    hipGetDevice(&deviceId);
    hipGetDeviceProperties(&prop, deviceId);

    int numberOfThreads = 256;
    int numberOfBlocks = 32 * prop.multiProcessorCount;

    Particle3D *h_a = NULL;
    Particle3D *h_b = NULL;

    hipHostMalloc((void**)&h_a, sizeof(Particle3D) * (num_particles / 2));
    hipHostMalloc((void**)&h_b, sizeof(Particle3D) * (num_particles / 2));
    hipHostMalloc((void**)&ha_forces, sizeof(float3) * (num_particles / 2));
    hipHostMalloc((void**)&hb_forces, sizeof(float3) * (num_particles / 2));

    Particle3D* d_a = NULL; 
    Particle3D* d_b = NULL; 

    hipMalloc((void**)&d_a, sizeof(Particle3D) * (num_particles / 2));
    hipMalloc((void**)&d_b, sizeof(Particle3D) * (num_particles / 2));
    hipMalloc((void**)&da_forces, sizeof(float3) * (num_particles / 2));
    hipMalloc((void**)&db_forces, sizeof(float3) * (num_particles / 2));

    hipStream_t stream[2];
    hipGraph_t graph[2];
    hipGraphExec_t instance[2];

    for (int i = 0; i < 2; i++) {
        hipStreamCreate(&stream[i]);
    }

    t_neighbourList *nb_list = nullptr;

    int pos = 0;    

    for (int i = 0; i < num_particles; ++i) {
        float x = fmod(pos, box_extension) ;
        float y = (pos >= box_extension) ? fmod(floor(pos * 2 / box_extension), box_extension): 0;
        float z = (pos * 4 >= box_extension * box_extension) ? fmod(floor((pos * 4) / (box_extension * box_extension) ), box_extension) : 0;

        if (i < (num_particles / 2)) {
            h_a[i] = Particle3D(float3{ x, y, z }, float3{ 0.0f, 0.0f, 0.0f }, 1.0f, 1.0f, nullptr, i);
            ha_forces[i] = float3{ 0.0f, 0.0f, 0.0f };
        } else {
            h_b[i - (num_particles / 2)] = Particle3D(float3{ x, y, z }, float3{ 0.0f, 0.0f, 0.0f }, 1.0f, 1.0f, nullptr, i);
            hb_forces[i - (num_particles / 2)] = float3{ 0.0f, 0.0f, 0.0f };
        }
        
        pos += 2;
    }

    std::cout << num_particles << ", ";

    hipMemcpyAsync(d_a, h_a, sizeof(Particle3D) * (num_particles / 2), hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(d_b, h_b, sizeof(Particle3D) * (num_particles / 2), hipMemcpyHostToDevice, stream[1]);

    hipMemcpyAsync(da_forces, ha_forces, sizeof(float3) * (num_particles / 2), hipMemcpyHostToDevice, stream[0]);
    hipMemcpyAsync(db_forces, hb_forces, sizeof(float3) * (num_particles / 2), hipMemcpyHostToDevice, stream[1]);

    for (int step = 0; step < time_steps; ++step) {
        hipMemset(da_forces, 0, sizeof(float3) * (num_particles / 2));
        hipMemset(db_forces, 0, sizeof(float3) * (num_particles / 2));

        // Begin capture for stream 0
        hipStreamBeginCapture(stream[0], hipStreamCaptureModeGlobal);

        compute_force_between_particles <<< numberOfBlocks, numberOfThreads, 0, stream[0]>>> (d_a, da_forces, num_particles, eps, sigma, k_n, gamma, gravity, box_extension, cut_off_radious, nb_list);
        apply_integrator_for_particle_euler <<< numberOfBlocks, numberOfThreads, 0, stream[0]>>> (d_a, da_forces, num_particles, step_size, box_extension);

        // End capture for stream 0
        hipStreamEndCapture(stream[0], &graph[0]);
        hipGraphInstantiate(&instance[0], graph[0], NULL, NULL, 0);

        // Begin capture for stream 1
        hipStreamBeginCapture(stream[1], hipStreamCaptureModeGlobal);

        compute_force_between_particles <<< numberOfBlocks, numberOfThreads, 0, stream[1]>>> (d_b, db_forces, num_particles, eps, sigma, k_n, gamma, gravity, box_extension, cut_off_radious, nb_list);
        apply_integrator_for_particle_euler <<< numberOfBlocks, numberOfThreads, 0, stream[1]>>> (d_b, db_forces, num_particles, step_size, box_extension);

        // End capture for stream 1
        hipStreamEndCapture(stream[1], &graph[1]);
        hipGraphInstantiate(&instance[1], graph[1], NULL, NULL, 0);

        for (int i = 0; i < 2; i++) {
            hipGraphLaunch(instance[i], stream[i]);
        }

        hipDeviceSynchronize();

        // Cleanup the graph instances after each iteration
        for (int i = 0; i < 2; i++) {
            hipGraphDestroy(graph[i]);
            hipGraphExecDestroy(instance[i]);
        }
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(da_forces);
    hipFree(db_forces);

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(ha_forces);
    hipHostFree(hb_forces);

    for (int i = 0; i < 2; i++) {
        hipStreamDestroy(stream[i]);
    }
}


*/

