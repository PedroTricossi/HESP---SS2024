#include "hip/hip_runtime.h"
#include <chrono>
#include <stdio.h>
#include <stdlib.h>

#include "../util.h"
#include "stream-util.h"

__global__ 
void stream(size_t nx, const double *__restrict__ src, double *__restrict__ dest) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < nx; i += stride)
        dest[i] = src[i] + 1;
}


int main(int argc, char *argv[]) {
    size_t nx, nItWarmUp, nIt, threadsPerBlock, numberOfBlocks, size;
    int deviceId, numberOfSMs;
    double* src;
    double* dest;

    parseCLA_1d(argc, argv, nx, nItWarmUp, nIt);
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    size = nx * sizeof(double);
    hipError_t r = hipMallocManaged(&src, size);
    hipError_t r2 = hipMallocManaged(&dest, size);

    if (r != hipSuccess) {
            fprintf(stderr, "CUDA Error on %s\n", hipGetErrorString(r));
            exit(0);
    }

    hipMemPrefetchAsync(src, size, deviceId);
    hipMemPrefetchAsync(dest, size, deviceId);

    threadsPerBlock = 256;
    numberOfBlocks = 32 * numberOfSMs;

    // init
    initStream(src, nx);

    // warm-up
    for (int i = 0; i < nItWarmUp; ++i) {
        stream<<<numberOfBlocks, threadsPerBlock>>>(nx, src, dest);
        // hipDeviceSynchronize();
        std::swap(src, dest);
    }
    hipDeviceSynchronize();

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < nIt; ++i) {
        stream<<<numberOfBlocks, threadsPerBlock>>>(nx, src, dest);
        // hipDeviceSynchronize();
        std::swap(src, dest);
        // hipDeviceSynchronize();
    }
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();

    printStats(end - start, nx, nIt, streamNumReads, streamNumWrites);

    // check solution
    checkSolutionStream(src, nx, nIt + nItWarmUp);

    hipFree(src);
    hipFree(dest);

    return 0;
}
